#include "hip/hip_runtime.h"
#include <stdio.h>
#include "logmin.h"

__global__ void logmin(int64_t *d_nums, size_t size) {
	int64_t idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (2*idx < size) {
		int64_t n1 = d_nums[2*idx];
		int64_t n2 = (2*idx+1 < size) ? d_nums[2*idx+1] : n1;

		d_nums[idx] = (n1 < n2) ? n1 : n2;
	}
}

int64_t run_logmin(int64_t* array, size_t array_len) {
	size_t array_size = sizeof(int64_t)*array_len;
	int64_t *h_nums = array;
	int64_t *final_nums = (int64_t*)malloc(array_size);

	int64_t *d_nums;
	hipError_t err = hipMalloc((void **)&d_nums, array_size);
	
	if (err != hipSuccess) {
		fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
					 __FILE__, __LINE__, hipGetErrorString( err) );		
	}

	err = hipMemcpy(d_nums, h_nums, array_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
					 __FILE__, __LINE__, hipGetErrorString( err) );		
	}

	// this blocksize and gridsize obviously does not work for all values of array_len
	size_t blocksize = 512;
	size_t gridsize = (array_len/blocksize)+1;
	for (int64_t size = array_len; size > 1; size = size%2+size/2) {
		logmin<<<gridsize,blocksize>>>(d_nums, size);

		err = hipPeekAtLastError();
		if (err != hipSuccess) {
			fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
						 __FILE__, __LINE__, hipGetErrorString( err) );		
		}

		err = hipMemcpy(final_nums, d_nums, array_size, hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			fprintf( stderr, "Cuda error in file '%s' in line %i : %s.\n",
						 __FILE__, __LINE__, hipGetErrorString( err) );		
		}
		// for (int64_t i = 0; i < size%2+size/2; i++) {
		// 	printf("%2d ", final_nums[i]);
		// }
		// printf("\n");
	}
	return final_nums[0];
}

int64_t run_linmin(int64_t *array, size_t array_size) {
	int64_t *nums = array;

	int64_t min = nums[0];
	for (size_t i = 0; i < array_size; i++) {
		min = (nums[i]<min) ? nums[i] : min;
	}
	return min;
}

int64_t *gen_array(size_t size) {
	int64_t *array = (int64_t*)malloc(sizeof(int64_t)*size);
	for (size_t i = 0; i < size; i++) {
		array[i] = 10289*(i+1)%20269;
	}
	return array;
}
